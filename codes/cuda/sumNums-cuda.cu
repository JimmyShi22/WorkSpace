#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

//1M
#define DATA_SIZE 1048576

#define THREAD_NUM 1024
#define BLOCK_NUM 32
int data[DATA_SIZE];

//产生大量0-9之间的随机数
void GenerateNumbers(int *number, int size)
{
    for (int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}

//打印设备信息
void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("---------------------------------------------------- \n");
    printf("Device Name : %s \n", prop.name);
    printf("totalGlobalMem : %zu MB\n",prop.totalGlobalMem/(1<<20));
    printf("sharedMemPerBlock : %zu B \n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d \n", prop.regsPerBlock);
    printf("warpSize : %d \n", prop.warpSize);
    printf("memPitch : %zu B \n", prop.memPitch);
    printf("maxThreadsPerBlock : %d \n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d \n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d \n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %zu B \n", prop.totalConstMem);
    printf("major.minor : %d.%d \n", prop.major, prop.minor);
    printf("clockRate : %d khz \n", prop.clockRate);
    printf("textureAlignment : %zu B \n", prop.textureAlignment);
    printf("deviceOverlap : %d \n", prop.deviceOverlap);
    printf("multiProcessorCount : %d \n", prop.multiProcessorCount);
    printf("---------------------------------------------------- \n");
}

//CUDA 初始化
bool InitCUDA()
{
    int count;

    //取得支持Cuda的装置的数目
    hipGetDeviceCount(&count);

    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;

    for (i = 0; i < count; i++) {

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        //打印设备信息
        printDeviceProp(prop);

        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(1);

    return true;
}


// __global__ 函数 (GPU上执行) 计算立方和
__global__ static void sumOfSquares(int *num, int* result, clock_t* time)
{

    //表示目前的 thread 是第几个 thread（由 0 开始计算）
    const int tid = threadIdx.x;

    int sum = 0;

    int i;

    //记录运算开始的时间
    clock_t start;

    //只在 thread 0（即 threadIdx.x = 0 的时候）进行记录
    if (tid == 0) start = clock();

    for (i = tid; i < DATA_SIZE; i += THREAD_NUM) {

        sum += num[i] * num[i] * num[i];

    }

    result[tid] = sum;

    //计算时间的动作，只在 thread 0（即 threadIdx.x = 0 的时候）进行
    if (tid == 0) *time = clock() - start;

}





int main()
{

    //CUDA 初始化
    if (!InitCUDA()) {
        return 0;
    }

    //生成随机数
    GenerateNumbers(data, DATA_SIZE);

    /*把数据复制到显卡内存中*/
    int* gpudata, *result;

    clock_t* time;

    //hipMalloc 取得一块显卡内存 ( 其中result用来存储计算结果，time用来存储运行时间 )
    hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
    hipMalloc((void**)&result, sizeof(int)*THREAD_NUM);
    hipMalloc((void**)&time, sizeof(clock_t));

    //hipMemcpy 将产生的随机数复制到显卡内存中
    //hipMemcpyHostToDevice - 从内存复制到显卡内存
    //hipMemcpyDeviceToHost - 从显卡内存复制到内存
    hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

    // 在CUDA 中执行函数 语法：函数名称<<<block 数目, thread 数目, shared memory 大小>>>(参数...);
    sumOfSquares << < 1, THREAD_NUM, 0 >> >(gpudata, result, time);


    /*把结果从显示芯片复制回主内存*/

    int sum[THREAD_NUM];

    clock_t time_use;

    //hipMemcpy 将结果从显存中复制回内存
    hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM, hipMemcpyDeviceToHost);
    hipMemcpy(&time_use, time, sizeof(clock_t), hipMemcpyDeviceToHost);

    //Free
    hipFree(gpudata);
    hipFree(result);
    hipFree(time);

    int final_sum = 0;

    for (int i = 0; i < THREAD_NUM; i++) {

        final_sum += sum[i];

    }

    printf("GPUsum: %d  gputime: %d\n", final_sum, time_use);

    final_sum = 0;

    for (int i = 0; i < DATA_SIZE; i++) {

        final_sum += data[i] * data[i] * data[i];

    }

    printf("CPUsum: %d \n", final_sum);

    return 0;
}
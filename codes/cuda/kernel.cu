#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdlib.h"
#include "stdio.h"

bool initCuda();
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b,clock_t* time)
{
    clock_t start = clock();
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
    *time = clock() - start;
}

int main()
{
    initCuda();
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}

void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("---------------------------------------------------- \n");
    printf("Device Name : %s \n", prop.name);
    printf("totalGlobalMem : %zu MB\n",prop.totalGlobalMem/(1<<20));
    printf("sharedMemPerBlock : %zu B \n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d \n", prop.regsPerBlock);
    printf("warpSize : %d \n", prop.warpSize);
    printf("memPitch : %zu B \n", prop.memPitch);
    printf("maxThreadsPerBlock : %d \n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d \n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d \n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %zu B \n", prop.totalConstMem);
    printf("major.minor : %d.%d \n", prop.major, prop.minor);
    printf("clockRate : %d khz \n", prop.clockRate);
    printf("textureAlignment : %zu B \n", prop.textureAlignment);
    printf("deviceOverlap : %d \n", prop.deviceOverlap);
    printf("multiProcessorCount : %d \n", prop.multiProcessorCount);
    printf("---------------------------------------------------- \n");
}
bool initCuda()
{
    hipError_t cudaStatus;

    int count;
    hipGetDeviceCount(&count);
    if(count == 0)
    {
        printf("There is no device \n");
        return false;
    }
    int i;
    for(i=0;i<count;++i)
    {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop,i) == hipSuccess)
        {
            if(prop.major>=1)
            {
                //打印设备信息
                printf("cuda device id: %d\n",i);
                printDeviceProp(prop);
                break;
            } 
        }
    }

    if(i==count) 
    {
        printf("There is no device supporting cuda 1.x\n");
        return false;
    }
    
    cudaStatus = hipSetDevice(0);
    return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;
    clock_t* time;
clock_t time_used;
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&time, sizeof(clock_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    // 在CUDA 中执行函数 语法：函数名称<<<block 数目, thread 数目, shared memory 大小>>>(参数...);
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b, time);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    printf("GPU time: %ld\n", time_used);

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


